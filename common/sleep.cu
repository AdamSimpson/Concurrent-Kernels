#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
 
static hipStream_t *streams;

// CUDA kernel to pause for at least num_cycle cycles
__global__ void sleep(int64_t num_cycles)
{
    int64_t cycles = 0;
    int64_t start = clock64();
    while(cycles < num_cycles) {
        cycles = clock64() - start;
    }
}
 
// Returns number of cycles required for requested seconds
extern "C" int64_t get_cycles(float seconds)
{
    // Get device frequency in KHz
    int64_t Hz;
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    Hz = int64_t(prop.clockRate) * 1000;
 
    // Calculate number of cycles to wait
    int64_t num_cycles;
    num_cycles = (int64_t)(seconds * Hz);
   
    return num_cycles;
}
 
// Create streams
extern "C" void create_streams(int num_streams)
{
    // Allocate streams
    streams = (hipStream_t *) malloc((num_streams+1)*sizeof(hipStream_t));    
 
    // Default stream
    streams[0] = NULL;

    // Primer kernel launch
    sleep<<< 1, 1 >>>(1); 

    // Create streams
    for(int i = 1; i <= num_streams; i++)
        hipStreamCreate(&streams[i]);
}
 
// Launches a kernel that sleeps for num_cycles
extern "C" void sleep_kernel(int64_t num_cycles, int stream_id)
{
    // Launch a single GPU thread to sleep
    int blockSize, gridSize;
    blockSize = 1;
    gridSize = 1;
 
    // Execute the kernel
    sleep<<< gridSize, blockSize, 0, streams[stream_id] >>>(num_cycles);
}
 
// Wait for stream to complete
extern "C" void wait_for_stream(int stream_id)
{
    hipStreamSynchronize(streams[stream_id]);
}
 
// Wait for streams to complete
extern "C" void wait_for_streams(int num_streams)
{
    for(int i = 1; i <= num_streams; i++)
        hipStreamSynchronize(streams[i]);
}
 
// Destroy stream objects
extern "C" void destroy_streams(int num_streams)
{
    // Clean up stream
    for(int i = 1; i <= num_streams; i++)
        hipStreamDestroy(streams[i]);
    free(streams);
}
