#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

// CUDA kernel to pause for at least num_cycle cycles
__global__ void sleep(int64_t num_cycles)
{
    int64_t cycles = 0;
    int64_t start = clock64();
    while(cycles < num_cycles) {
        cycles = clock64() - start;
    }
}

// Returns number of cycles required for requested seconds
extern "C" int64_t get_cycles(float seconds)
{
    // Get device frequency in KHz
    int64_t Hz;
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    Hz = int64_t(prop.clockRate) * 1000;

    // Calculate number of cycles to wait
    int64_t num_cycles;
    num_cycles = (int64_t)(seconds * Hz);
  
    return num_cycles;
}

// Launches a kernel that sleeps for num_cycles
extern "C" void sleep_kernel(int64_t num_cycles)
{
    // Our kernel will launch a single thread to sleep the kernel
    int blockSize, gridSize;
    blockSize = 1;
    gridSize = 1;

    // Execute the kernel in default stream
    sleep<<< gridSize, blockSize >>>(num_cycles);
}

// Wait for all work  to complete
extern "C" void wait_for_gpu()
{
    hipDeviceSynchronize();
}
