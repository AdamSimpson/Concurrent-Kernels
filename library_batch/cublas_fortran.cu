
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "hipblas.h"

extern "C" int f_cublasCreate(hipblasHandle_t **handle)
{
    *handle = (hipblasHandle_t*)malloc(sizeof(hipblasHandle_t));
    return hipblasCreate(*handle);
}

extern "C" int f_cublasDgemm(hipblasHandle_t *handle,
               hipblasOperation_t transa, hipblasOperation_t transb, 
              int m, int n, int k, 
              const double *alpha,
              const double *A, int lda, 
              const double *B, int ldb,
              const double *beta, 
              double *C, int ldc)
{
    return hipblasDgemm(*handle,transa,transb,m,n,k,alpha,A,lda,B,ldb,beta,C,ldc);
}

extern "C" void f_cublasDestroy(hipblasHandle_t *handle_ptr)
{
    hipblasDestroy(*handle_ptr);
    free(handle_ptr);
}

extern "C" int f_cudaStreamCreate(hipStream_t **stream)
{
    *stream = (hipStream_t *) malloc(sizeof(hipStream_t));
    hipStreamCreate(*stream);
}

